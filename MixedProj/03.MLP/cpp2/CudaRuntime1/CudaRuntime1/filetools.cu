
#include <hip/hip_runtime.h>

void saveFloatsToFile(char* filename, float* floats, int arySize) {
    FILE* fb;
    fb = fopen( (char*) filename, "wb");
    for (int i = 0; i < arySize; i++) {
        fwrite(&floats[i], sizeof(float), 1, fb);
    }
    fclose(fb);
}


void loadFloatsToFile(char* filename, float* floats, int arySize) {
    FILE* fb;
    fb = fopen((char*)filename, "rb");
    for (int i = 0; i < arySize; i++) {
        fread(&floats[i], sizeof(float), 1, fb);
    }
    fclose(fb);
}

void testLoadAndSave() {
    float* Test = new float[10];
        float* Check = new float[10];
        Test[0] = -.5f;
        Test[1] = 17.17f;
        Test[2] = .00000001f;
        Test[3] = 125.0f;
        Test[4] = -9999.0f;
        Test[5] = -99.0005f;

        saveFloatsToFile((char*)"testFile", Test, 5);
        loadFloatsToFile((char*)"testFile", Check, 5);
        printf("\n IN: %f : %f : %f : %f : %f ", Test[0], Test[1], Test[2], Test[3], Test[4]);
        printf("\nOUT: %f : %f : %f : %f : %f ", Check[0], Check[1], Check[2], Check[3], Check[4]);
}