#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

 
#include <iostream>
#include <ctime>
#include <fstream>
#include <string.h>
#include <vector>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t mullAndaddFloatWithCuda(float* c, const float* a, const float* b, unsigned int size);
void saveFloatsToFile(char* filename, float* floats, int size);
void loadFloatsToFile(char* filename, float* floats, int arySize);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void mullFloatArrays(float* c, const float* a, const float* b)
{
    int i = threadIdx.x;
    //c[i] = a[i] * b[i]; 
    c[i] = 1.0/(1.0+ exp ( a[i]+b[i]));
    //c[i] = .1;
}

__global__ void sumOfC(float* d, float* c)
{
    int i = threadIdx.x;
    if (i % 2 == 0) { d[i/2] = c[i] + c[i + 1];  c[i + 1] = 0; c[i] = 0; }  
}

int main()
{
    int const percent = 1; // 50 0.010; // change to 50 !
    int const len = percent; // *100;
    int const lenx = len * 6;
    
    int const IMGSIZE = 784; // ; 28 * 28
    int const Lay1out = 64; // neurons = layer out numbers

    //printf( "#  --- C++ ---\n+lenx:%i", lenx);
    srand((int)time(0));

    float** X = new float* [lenx];
    for (int i = 0; i < lenx; i++) {
        X[i] = new float[ IMGSIZE ];
    }

    float** W = new float* [ 64 ];
    for (int i = 0; i < Lay1out; i++) {
        W[i] = new float[ IMGSIZE ];
        for (int j = 0; j < IMGSIZE; j++) {
            W[i][j] = -1.0f + (rand() % 1000) / 500.0f;
        }
    }

    uint8_t* Y = new uint8_t[lenx];


    
    ifstream  inputFileStreamX("../../../data/train-images-idx3-ubyte", ios::in | ios::binary); // 16, percent, 6)
    ifstream  inputFileStreamY("../../../data/train-labels-idx1-ubyte", ios::in | ios::binary); //  8, percent, 6)


    inputFileStreamX.ignore(16 * sizeof(uint8_t));
    inputFileStreamY.ignore(8 * sizeof(uint8_t));
    uint8_t* tmp = new uint8_t[784];
    for (int i = 0; i < lenx; i++) {
        inputFileStreamX.read((char*)tmp, 784 * sizeof(char));
        for (int j = 0; j < 784; j++) {
            X[i][j] = tmp[j]/255.0f;
        }
        inputFileStreamY.read((char*)&Y[i],  sizeof(char));
    }

    inputFileStreamX.close();
    inputFileStreamY.close();

//    for (int i = 0; i < 50; i++) {
//        printf( "%i: %f\n", Y[i], X[i][300+i]);
//    }
    

//    if ( true ) return 0;

//    vector<double>* Z = new vector<double>[lenx];
//    vector<double>* X1 = new vector<double>[lenx];
//    vector<double>* Y1 = new vector<double>[lenx];
    //const int arySize = 8;
    //float X_[arySize] = { 0.1f, 0.2f, 0.3f, 0.4f, 0.5f, 0.6f, 0.7f, 0.8f }; //new float [8];
    //float W_ [arySize] = { -0.4f, 0.2f, -.3f, .3f, -.1f, .5f, -.2f, .4f };
    //float* W_ = new float[arySize]; // null; { -0.4f, 0.2f, -.3f, .3f, -.1f, .5f, -.2f, .4f }; //new float [8];
    float* YLay1 = new float[Lay1out];
    float* D = new float[IMGSIZE];
    float* C = new float[IMGSIZE];
     

    //saveFloatsToFile((char*)"Weights.bin", W_, arySize);
    //loadFloatsToFile((char*)"Weights.bin", W_, arySize);
    //printf( "\n->%f",W_[0]);


    //-- start
    clock_t before = clock();

    hipError_t cudaStatus1 = mullAndaddFloatWithCuda( C, X[0], W[0], IMGSIZE); // X[0] - first X, W[0] - weights first neutron = first Y

    clock_t duration = clock() - before;
    printf("\r\nduration: %d [clocks tick], %d[sek]\r\n", duration, duration/CLOCKS_PER_SEC );

    printf ( "\nC[14*28+14]:%f, X[0][14*28+14]%f, W[0][14*28+14]%f\n", C[14 * 28 + 14], X[0][14*28+14], W[0][14 * 28 + 14]);
    printf("Y[0]:%f \n", C[0]);

    cudaStatus1 = hipDeviceReset();
    if (cudaStatus1 != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    exit(0);
    return 0;
   



    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };



    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}





// Helper function for using CUDA to add vectors in parallel.
hipError_t mullAndaddFloatWithCuda(float* c, const float* a, const float* b, unsigned int size)
{
    // size = 88; 88 is OK, 89 throw error !
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;
    float* dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc( (void**)&dev_c, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**) &dev_d, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // Launch a kernel on the GPU with one thread for each element.
    mullFloatArrays <<< 1, size >> > (dev_c, dev_a, dev_b);
    hipDeviceSynchronize();

    // log(2) 784 < log(2) 1024 = 10 circles
    for (int i = 0; i < 5; i++) {
        sumOfC <<< 1, size >> > ( dev_d, dev_c );
        //hipDeviceSynchronize(); 
        sumOfC <<< 1, size >> > ( dev_c, dev_d );
        //hipDeviceSynchronize(); 
    }



    //dev_c[0] = dev_d[0];

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
     

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    return cudaStatus;

Error:
    hipFree(dev_d);
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}


















// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<< 1, size >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}



void saveFloatsToFile(char* filename, float* floats, int arySize) {
    FILE* fb;
    fb = fopen( (char*) filename, "wb");
    for (int i = 0; i < arySize; i++) {
        fwrite(&floats[i], sizeof(float), 1, fb);
    }
    fclose(fb);
}


void loadFloatsToFile(char* filename, float* floats, int arySize) {
    FILE* fb;
    fb = fopen((char*)filename, "rb");
    for (int i = 0; i < arySize; i++) {
        fread(&floats[i], sizeof(float), 1, fb);
    }
    fclose(fb);
}