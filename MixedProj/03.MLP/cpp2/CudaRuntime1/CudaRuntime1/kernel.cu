﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <iostream>
#include <ctime>
#include <fstream>
#include <string.h>
#include <vector>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
   int const percent = 50;
    long const len = percent * 100;

    printf( "#  --- C++ ---\n");
    double X[len * 784 * 6];
    double Y[len * 1 * 6];
    
    //ifstream  inputFileStreamX("data/train-images-idx3-ubyte", ios::in | ios::binary); // 16, percent, 6)
    //ifstream  inputFileStreamY("data/train-labels-idx1-ubyte", ios::in | ios::binary); //  8, percent, 6)

    
    //inputFileStreamX.ignore(16 * sizeof(double));
    //inputFileStreamY.ignore(8 * sizeof(double));


    //inputFileStreamX.read((char*)&X, percent * 100 * 784 * 6 * sizeof(double));
    //inputFileStreamY.read((char*)&Y, percent * 100 * 1 * 6 * sizeof(double));


    //inputFileStreamX.close();
    //inputFileStreamY.close();
 
    printf("%s", X);










    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<< 1, size >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


void loadFileX() {

}











/*


//#include <dirent.h>



int main() {






   cout << X;

   exit(0);


//-- start
  clock_t before = clock();

   double w1=0.0;
   double w0=0.0;

   double xsr=0.0;
   double ysr=0.0;


  for (int c = 0; c < CYCLES; c++) {

          xsr = 0.0;
          ysr = 0.0;
          w1  = 0.0;
          w0  = 0.0;


      for ( int i=0; i<len; i++ ){
         xsr +=  X[i];
         ysr +=  Y[i];
      }

   xsr=xsr / len;
   ysr=ysr / len;


   double sumTop=0.0;
   double sumBottom=0.0;

      for ( int i=0;i<len;i++ ){ //  xtmp = X[i]-sr ! ;
       sumTop   += ((X[i]-xsr)*(Y[i]-ysr));
      sumBottom += ((X[i]-xsr)*(X[i]-xsr));
      }
      w1 = sumTop / sumBottom;
      w0 = ysr -(w1 * xsr) ;
  }

  clock_t duration = clock() - before;
  cout << "#  X[" << percent << "] * " << CYCLES << "\n";
  cout << "#  time: " << (float)duration / CLOCKS_PER_SEC << " [sek.],  w0: " << w1 << ", w1: " << w0 << "\n";
  cout << "\nc[]=" << (float)duration / CLOCKS_PER_SEC << "\n" ;



  return 0;
}






/*

//   printf( " %E %E %E\r\n", X[0], X[1], X[2]);
//   printf( " %E %E %E\r\n", Y[0], Y[1], Y[2]);
len = strlen(name);
dirp = opendir(".");
while ((dp = readdir(dirp)) != NULL)
        if (dp->d_namlen == len && !strcmp(dp->d_name, name)) {
                (void)closedir(dirp);
                return FOUND;
        }
(void)closedir(dirp);
return NOT_FOUND;

*/




//https://www.w3schools.com/c/c_ref_stdlib.php
/*

    FILE *f;
    f = fopen("mainfinal.c" , "r");
    fseek(f, 0, SEEK_END);
    unsigned long len = (unsigned long)ftell(f);
    printf("%ld\n", len);
    fclose(f);
*/



//   while ( (file = readdir(dirp)) != NULL ){
//      if ( fileName.compare(   file->d_name ) != 0) { continue; }
//           file->seekg(0, ios::end);
//   ifstream inputFileStream( "data/" + fileName , ios::in | ios::binary );

   //ifstream.seekg (0, ios::end);
   //len = is.tellg();
    //cout << "Size: " << len << " bytes";


//    }
//closedir( dirp );


/*


epochs = 100

num_classes = 10


def readFileX ( fileName , offset, percent, multi ):
    file=open( fileName, 'rb' )
    file.read( offset )
    data=np.fromfile( fileName, np.uint8, percent*100*784*multi, '', offset )
    data=data.reshape(percent*100*multi, 784)
    data=(data/255)
    file.close()
    return data

def readFileY ( fileName , offset, percent, multi ):
    file=open( fileName, 'rb' )
    file.read( offset )
    len=percent*100*multi
    data=np.fromfile( fileName, np.uint8, len, '', offset )
    file.close()
    return data



trainX = readFileX ('data/train-images-idx3-ubyte', 16, percent ,6 )
trainY = readFileY ('data/train-labels-idx1-ubyte', 8, percent, 6 )
testX = readFileX ('data/t10k-images-idx3-ubyte', 16, percent, 1  )
testY = readFileY ('data/t10k-labels-idx1-ubyte', 8, percent, 1 )


trainX = trainX.astype("float32") # / 255
testX = testX.astype("float32") # / 255
trainX = trainX.reshape(6*percent*100, 784).astype("float32") / 255
testX = testX.reshape(1*percent*100, 784).astype("float32") / 255



model = tf.keras.models.Sequential([
  tf.keras.layers.Input(shape=(784,)),
  tf.keras.layers.Dense(64, activation='sigmoid'),
  tf.keras.layers.Dense(64, activation='sigmoid'),
  tf.keras.layers.Dropout(0.2),
  tf.keras.layers.Dense(10, activation='softmax')
])

model.compile(optimizer='adam',
  loss='sparse_categorical_crossentropy',
  metrics=['accuracy'])

start=time.time()


with tf.device('/device:GPU:0'):
   model.fit(trainX, trainY, epochs=epochs, verbose=0)

end=time.time()
d=end-start

clear_session()


print("# Python Tensorflow Time: " , d)

*/