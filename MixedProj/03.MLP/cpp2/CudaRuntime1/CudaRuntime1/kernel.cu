#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <ctime>
#include <fstream>
#include <string.h>
#include <vector>

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int const percent = 0.010; // change to 50 !
    int const len = percent * 100;
    int const lenx = len * 6;

    printf( "#  --- C++ ---\n+lenx:%i", lenx);

    double** X = new double * [lenx];
    for (int i = 0; i < lenx; i++) {
        X[i] = new double[28*28];
    }
    uint8_t* Y = new uint8_t[lenx];

    
    ifstream  inputFileStreamX("../../../data/train-images-idx3-ubyte", ios::in | ios::binary); // 16, percent, 6)
    ifstream  inputFileStreamY("../../../data/train-labels-idx1-ubyte", ios::in | ios::binary); //  8, percent, 6)

    
    inputFileStreamX.ignore(16 * sizeof(uint8_t));
    inputFileStreamY.ignore(8 * sizeof(uint8_t));
    uint8_t* tmp = new uint8_t[784];
    for (int i = 0; i < lenx; i++) {
        inputFileStreamX.read((char*)tmp, 784 * sizeof(char));
        for (int j = 0; j < 784; j++) {
            X[i][j] = tmp[j]/255.0f;
        }
        inputFileStreamY.read((char*)&Y[i],  sizeof(char));
    }

    inputFileStreamX.close();
    inputFileStreamY.close();

//    for (int i = 0; i < 50; i++) {
//        printf( "%i: %f\n", Y[i], X[i][300+i]);
//    }
    

//    if ( true ) return 0;

//    vector<double>* Z = new vector<double>[lenx];
//    vector<double>* X1 = new vector<double>[lenx];
//    vector<double>* Y1 = new vector<double>[lenx];
    double** W = new double* [64];

    hipError_t cudaStatus = addWithCuda(X[0], , Z, lenx);






    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };



    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<< 1, size >>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
